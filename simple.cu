#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "header.h"
#include "util.cuh"
#include "util.h"

#define BLOCK_SMALL		8
#define	BLOCK_LARGE		16
#define	THREAD_Z		8

__global__ void gpu_simple_stencil_kernel(
	global_const_t *g,	// i:
	double *q,			// i:
	double *flux		// o:
){
	bool compute=false;
	int z, zidx, idx,out,si,sj,sk,tidx,tidy;
	double unp1, unp2, unp3, unp4, unm1, unm2, unm3, unm4;
	double flux_irho;

	__shared__ double      s_qu[BLOCK_LARGE+NG+NG][BLOCK_LARGE+NG+NG];

	// Load to shared mem
	for(z=0;z<THREAD_Z;z++){

		si = blockIdx.x*blockDim.x+threadIdx.x;
		sj = blockIdx.y*blockDim.y+threadIdx.y;
		sk = (blockIdx.z*blockDim.z+threadIdx.z)*THREAD_Z + z;

		out = sk*g->plane_offset_padded + sj*g->pitch[0] + si;
		compute = (si < g->dim[0] && sj < g->dim[1] && sk < g->dim[2]);


		__syncthreads();	//for the next z round
		for(sj=blockIdx.y*blockDim.y+threadIdx.y, tidy=threadIdx.y; tidy < BLOCK_LARGE+NG+NG; sj+=blockDim.y, tidy+=blockDim.y){
			for(si=blockIdx.x*blockDim.x+threadIdx.x, tidx=threadIdx.x; tidx < BLOCK_LARGE+NG+NG; si+=blockDim.x, tidx+=blockDim.x){
				if( si < g->dim_g[0] && sj < g->dim_g[1] && sk < g->dim_g[2]){

					idx = (sk+g->ng)*g->plane_offset_g_padded + sj*g->pitch_g[0] + si;
					s_qu[tidy][tidx] = q[idx + qu*g->comp_offset_g_padded];

				}
			}
		}
		__syncthreads();

		if(compute){

	#define	s_qu(i)			s_qu[threadIdx.y+g->ng][threadIdx.x+g->ng+(i)]

			flux_irho = - ( g->ALP*(s_qu(1)-s_qu(-1))
						  + g->BET*(s_qu(2)-s_qu(-2))
						  + g->GAM*(s_qu(3)-s_qu(-3))
						  + g->DEL*(s_qu(4)-s_qu(-4)))*g->dxinv[0];

			flux_irho -=  ( g->ALP*(s_qu(1)-s_qu(-1))
						  + g->BET*(s_qu(2)-s_qu(-2))
						  + g->GAM*(s_qu(3)-s_qu(-3))
						  + g->DEL*(s_qu(4)-s_qu(-4)))*g->dxinv[1];

	#undef	s_qu

		}

		/** Z dimension **/
		si = blockIdx.x*blockDim.x+threadIdx.x;
		sj = blockIdx.y*blockDim.y+threadIdx.y;
		if(compute){
			idx = (sk+g->ng)*g->plane_offset_g_padded + (sj+g->ng)*g->pitch_g[0] + si+g->ng;

			unp1 = q[qu*g->comp_offset_g_padded + idx + 1*g->plane_offset_g_padded];
			unp2 = q[qu*g->comp_offset_g_padded + idx + 2*g->plane_offset_g_padded];
			unp3 = q[qu*g->comp_offset_g_padded + idx + 3*g->plane_offset_g_padded];
			unp4 = q[qu*g->comp_offset_g_padded + idx + 4*g->plane_offset_g_padded];
			unm1 = q[qu*g->comp_offset_g_padded + idx - 1*g->plane_offset_g_padded];
			unm2 = q[qu*g->comp_offset_g_padded + idx - 2*g->plane_offset_g_padded];
			unm3 = q[qu*g->comp_offset_g_padded + idx - 3*g->plane_offset_g_padded];
			unm4 = q[qu*g->comp_offset_g_padded + idx - 4*g->plane_offset_g_padded];


			flux_irho -=  ( g->ALP*(unp1-unm1)
						  + g->BET*(unp2-unm2)
						  + g->GAM*(unp3-unm3)
						  + g->DEL*(unp4-unm4))*g->dxinv[2];

			// Update global memory
			flux[out + irho*g->comp_offset_padded] = flux_irho;

		}
	}
}

__global__ void usleep(){
	clock_t start = clock();
	clock_t now = clock();
	clock_t cycles;

	do{
		cycles = (now > start)? (now-start):(now+(0xffffffff - start));
		now = clock();

	}while(cycles < CLOCKS_PER_SEC);
}

void gpu_simple_stencil(
	global_const_t h_const, 	// i: Global struct containing application parameters
	global_const_t *d_const,	// i: Device pointer to global struct containing application paramters
	double *d_q,				// i:
	double *d_flux				// o:
){
	// Set preferred cache configuration (48KB smem | 16KB smem)
	// hipFuncCachePreferShared | hipFuncCachePreferL1
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	dim3 block_dim(BLOCK_LARGE, BLOCK_LARGE, 1);
	dim3 grid_dim(CEIL(h_const.dim[0], BLOCK_LARGE), CEIL(h_const.dim[1], BLOCK_LARGE), CEIL(h_const.dim[2], THREAD_Z));


	struct timeval s, e;
	gettimeofday(&s, NULL);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	gpu_simple_stencil_kernel<<<grid_dim, block_dim>>>(d_const, d_q, d_flux);
//	usleep<<<grid_dim, block_dim>>>();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipDeviceSynchronize();
	gettimeofday(&e, NULL);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("%lf vs %lf\n", elapsedTime, (double)(e.tv_sec-s.tv_sec) + 1.0E-6*(e.tv_usec-s.tv_usec));

	hipEventDestroy(start);
	hipEventDestroy(stop);
}
