#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "header.h"
#include "util.h"

#define BLOCK_DIM	16

__global__ void gpu_diffterm_lv3_kernel(
	global_const_t *g,			// i: Global struct containing application parameters
	double *q,					// i:
	double *difflux,			// o:
	double *flux				// o: set zeroes for hypterm
){
	int si, sj, sk, idx, idx_g;
	double mechwork;

	si = blockIdx.x*blockDim.x + threadIdx.x;
	sj = blockIdx.y*blockDim.y + threadIdx.y;
	sk = blockIdx.z;

	idx = sk*g->plane_offset_padded + sj*g->pitch[0] + si;
	idx_g	= (sk+g->ng)*g->plane_offset_g_padded + (sj+g->ng)*g->pitch_g[0] + si+g->ng;
	if(si < g->dim[0] && sj < g->dim[1] && sk < g->dim[2]){

		mechwork = 	difflux[idx + iene*g->comp_offset_padded] +
					difflux[idx + imz*g->comp_offset_padded]*q[idx_g + qw*g->comp_offset_g_padded];
		difflux[idx + iene*g->comp_offset_padded] =
					g->alam*(g->temp[TXX][idx]+g->temp[TYY][idx]+g->temp[TZZ][idx]) + mechwork;
	}
}

void gpu_diffterm2(
	global_const_t h_const, 	// i: Global struct containing application parameters
	global_const_t *d_const,	// i: Device pointer to global struct containing application paramters
	double *d_q,				// i:
	double *d_difflux,			// o:
	double *d_flux				// o: just set zeroes for hypterm
){
	kernel_const_t h_kc;
	dim3 grid_dim(CEIL(h_const.dim_g[0], BLOCK_DIM), CEIL(h_const.dim_g[1], BLOCK_DIM), h_const.dim_g[2]);
	dim3 block_dim(BLOCK_DIM, BLOCK_DIM);

	gpu_diffterm_lv1_lv2(h_const, d_const, d_q, d_difflux, d_flux);
	gpu_diffterm_lv3_kernel<<<grid_dim, block_dim>>>(d_const, d_q, d_difflux, d_flux);
}
